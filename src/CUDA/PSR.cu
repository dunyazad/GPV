#include "hip/hip_runtime.h"
#include "PSR.cuh"

#include <vtkHeaderFiles.h>

#include <App/Utility.h>

#include <Debugging/VisualDebugging.h>
using VD = VisualDebugging;

#include <Algorithm/MarchingCubes.hpp>

namespace CUDA
{
	namespace PSR
	{
#pragma region PatchBuffers
		PatchBuffers::PatchBuffers(int width, int height)
			: width(width), height(height),
			inputPoints(thrust::device_vector<Eigen::Vector3f>(width* height)),
			inputNormals(thrust::device_vector<Eigen::Vector3f>(width* height)),
			inputColors(thrust::device_vector<Eigen::Vector3f>(width* height))
		{
		}

		PatchBuffers::~PatchBuffers()
		{
		}

		void PatchBuffers::Clear()
		{
			numberOfInputPoints = 0;
			Eigen::Vector3f zeroVector(0.0f, 0.0f, 0.0f);
			thrust::fill(inputPoints.begin(), inputPoints.end(), zeroVector);
			thrust::fill(inputNormals.begin(), inputNormals.end(), zeroVector);
			thrust::fill(inputColors.begin(), inputColors.end(), zeroVector);
		}

		void PatchBuffers::FromPLYFile(const PLYFormat& ply)
		{
			Clear();

			numberOfInputPoints = ply.GetPoints().size() / 3;

			hipMemcpy(thrust::raw_pointer_cast(inputPoints.data()),
				ply.GetPoints().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			hipMemcpy(thrust::raw_pointer_cast(inputNormals.data()),
				ply.GetNormals().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			hipMemcpy(thrust::raw_pointer_cast(inputColors.data()),
				ply.GetColors().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			//for (size_t i = 0; i < ply.GetPoints().size() / 3; i++)
			//{
			//	auto px = ply.GetPoints()[i * 3 + 0];
			//	auto py = ply.GetPoints()[i * 3 + 1];
			//	auto pz = ply.GetPoints()[i * 3 + 2];

			//	auto nx = ply.GetNormals()[i * 3 + 0];
			//	auto ny = ply.GetNormals()[i * 3 + 1];
			//	auto nz = ply.GetNormals()[i * 3 + 2];

			//	auto cx = ply.GetColors()[i * 3 + 0];
			//	auto cy = ply.GetColors()[i * 3 + 1];
			//	auto cz = ply.GetColors()[i * 3 + 2];

			//	auto c4 = Color4::FromNormalized(cx, cy, cz, 1.0f);
			//	VD::AddSphere("points", { px, py, pz }, { 0.05f, 0.05f, 0.05f }, { nx, ny, nz }, c4);
			//}
		}
#pragma endregion

		struct Voxel
		{
			Eigen::Vector3f normal;
			int weight;
			__host__ __device__ Voxel() : normal(0.0f, 0.0f, 0.0f), weight(0) {}
		};

		__host__ __device__
			uint64_t GetMortonCode(
				const Eigen::Vector3f& min,
				const Eigen::Vector3f& max,
				int maxDepth,
				const Eigen::Vector3f& position) {
			// Validate and compute range
			Eigen::Vector3f range = max - min;
			range = range.cwiseMax(Eigen::Vector3f::Constant(1e-6f)); // Avoid zero range

			// Normalize position
			Eigen::Vector3f relativePos = (position - min).cwiseQuotient(range);

			// Clamp to [0, 1]
			relativePos = relativePos.cwiseMax(0.0f).cwiseMin(1.0f);

			// Scale to Morton grid size
			uint32_t maxCoordinateValue = (1 << maxDepth) - 1; // maxCoordinateValue = 1 for maxDepth = 1
			uint32_t x = static_cast<uint32_t>(roundf(relativePos.x() * maxCoordinateValue));
			uint32_t y = static_cast<uint32_t>(roundf(relativePos.y() * maxCoordinateValue));
			uint32_t z = static_cast<uint32_t>(roundf(relativePos.z() * maxCoordinateValue));

			// Compute Morton code
			uint64_t mortonCode = 0;
			for (int i = 0; i < maxDepth; ++i) {
				mortonCode |= ((x >> i) & 1ULL) << (3 * i);
				mortonCode |= ((y >> i) & 1ULL) << (3 * i + 1);
				mortonCode |= ((z >> i) & 1ULL) << (3 * i + 2);
			}

			return mortonCode;
		}

		__host__ __device__
			uint3 GetIndex(const Eigen::Vector3f& gridCenter, uint3 gridDimensions, float voxelSize, const Eigen::Vector3f& position)
		{
			Eigen::Vector3f halfGridSize = Eigen::Vector3f(
				(float)gridDimensions.x * voxelSize * 0.5f,
				(float)gridDimensions.y * voxelSize * 0.5f,
				(float)gridDimensions.z * voxelSize * 0.5f
			);

			Eigen::Vector3f gridMin = gridCenter - halfGridSize;
			Eigen::Vector3f relativePosition = position - gridMin;

			uint3 index = make_uint3(UINT_MAX, UINT_MAX, UINT_MAX);

			if (relativePosition.x() < 0.0f || relativePosition.x() >= (float)gridDimensions.x * voxelSize ||
				relativePosition.y() < 0.0f || relativePosition.y() >= (float)gridDimensions.y * voxelSize ||
				relativePosition.z() < 0.0f || relativePosition.z() >= (float)gridDimensions.z * voxelSize)
			{
				return index;
			}
			else
			{
				index.x = (uint32_t)floorf(relativePosition.x() / voxelSize);
				index.y = (uint32_t)floorf(relativePosition.y() / voxelSize);
				index.z = (uint32_t)floorf(relativePosition.z() / voxelSize);
			}

			return index;
		}

		__host__ __device__
			Eigen::Vector3f GetPosition(const Eigen::Vector3f& gridCenter, uint3 gridDimensions, float voxelSize, const uint3& index)
		{
			Eigen::Vector3f halfGridSize = Eigen::Vector3f(
				(float)gridDimensions.x * voxelSize * 0.5f,
				(float)gridDimensions.y * voxelSize * 0.5f,
				(float)gridDimensions.z * voxelSize * 0.5f
			);

			Eigen::Vector3f gridMin = gridCenter - halfGridSize;

			// Calculate the position of the given voxel using the provided index
			Eigen::Vector3f position = Eigen::Vector3f(
				gridMin.x() + (float)index.x * voxelSize/* + voxelSize * 0.5f*/,
				gridMin.y() + (float)index.y * voxelSize/* + voxelSize * 0.5f*/,
				gridMin.z() + (float)index.z * voxelSize/* + voxelSize * 0.5f*/
			);

			return position;
		}

		__host__ __device__
			size_t GetFlatIndex(const uint3& index, const uint3& dimensions) {
			if (index.x >= dimensions.x || index.y >= dimensions.y || index.z >= dimensions.z) {
				return UINT_MAX;
			}
			return index.z * dimensions.x * dimensions.y + index.y * dimensions.x + index.x;
		}

		__host__ __device__
			bool isBoundary(uint32_t x, uint32_t y, uint32_t z, const uint3& dimensions) {
			return (x == 0 || y == 0 || z == 0 ||
				x == dimensions.x - 1 ||
				y == dimensions.y - 1 ||
				z == dimensions.z - 1);
		}

		__host__ __device__
			void matVecMul(const std::vector<float>& A, const std::vector<float>& x, std::vector<float>& b, int size) {
			for (int i = 0; i < size; ++i) {
				b[i] = 0;
				for (int j = 0; j < size; ++j) {
					b[i] += A[i * size + j] * x[j];
				}
			}
		}

		// ���� �ع�: ���콺 �ҰŹ�
		__host__ __device__
			void gaussSolve(float* A, float* x, float* b, int size) {
			for (int i = 0; i < size; ++i) {
				// �밢�� ��Ҹ� 1�� ����
				float diag = A[i * size + i];
				for (int j = 0; j < size; ++j) {
					A[i * size + j] /= diag;
				}
				b[i] /= diag;

				// �ٸ� �� ����
				for (int k = i + 1; k < size; ++k) {
					float factor = A[k * size + i];
					for (int j = 0; j < size; ++j) {
						A[k * size + j] -= factor * A[i * size + j];
					}
					b[k] -= factor * b[i];
				}
			}

			// ������ ����
			for (int i = size - 1; i >= 0; --i) {
				x[i] = b[i];
				for (int j = i + 1; j < size; ++j) {
					x[i] -= A[i * size + j] * x[j];
				}
			}
		}

		void TestPSR()
		{
			auto t = Time::Now();

			stringstream ss;
			ss << "C:\\Resources\\3D\\PLY\\Complete\\Lower_pointcloud.ply";

			PLYFormat ply;
			ply.Deserialize(ss.str());
			cout << "ply min : " << ply.GetAABB().min().transpose() << endl;
			cout << "ply max : " << ply.GetAABB().max().transpose() << endl;

			t = Time::End(t, "Load ply");

			PatchBuffers patchBuffers(ply.GetPoints().size() / 3, 1);
			patchBuffers.FromPLYFile(ply);

			t = Time::End(t, "Copy data to device");

			//Eigen::Vector3f total_min(-17.5f, -17.5f, -17.5f);
			//Eigen::Vector3f total_max(17.5f, 17.5f, 17.5f);
			Eigen::Vector3f total_min(-10.0f, -10.0f, -10.0f);
			Eigen::Vector3f total_max(10.0f, 10.0f, 10.0f);
			Eigen::Vector3f total_diff = total_max - total_min;
			Eigen::Vector3f total_center = (total_max + total_min) * 0.5f;
			float voxelSize = 0.1f;

			uint3 total_dimensions;
			total_dimensions.x = (uint32_t)ceilf(total_diff.x() / voxelSize);
			total_dimensions.y = (uint32_t)ceilf(total_diff.y() / voxelSize);
			total_dimensions.z = (uint32_t)ceilf(total_diff.z() / voxelSize);

			thrust::device_vector<Voxel> volume(total_dimensions.x / 2 * total_dimensions.y / 2 * total_dimensions.z / 2);
			auto d_volume = thrust::raw_pointer_cast(volume.data());
			thrust::device_vector<float> divergences(total_dimensions.x / 2 * total_dimensions.y / 2 * total_dimensions.z / 2);
			auto d_divergences = thrust::raw_pointer_cast(divergences.data());
			thrust::device_vector<float> potentials(total_dimensions.x / 2 * total_dimensions.y / 2 * total_dimensions.z / 2);
			auto d_potentials = thrust::raw_pointer_cast(potentials.data());

			for (size_t i = 0; i < 8; i++)
			{
				float minX = 0.0f;
				float maxX = 0.0f;
				float minY = 0.0f;
				float maxY = 0.0f;
				float minZ = 0.0f;
				float maxZ = 0.0f;

				if (i & 0b001)
				{
					minX = total_center.x();
					maxX = total_max.x();
				}
				else
				{
					minX = total_min.x();
					maxX = total_center.x();
				}
				if (i & 0b010)
				{
					minY = total_center.y();
					maxY = total_max.y();
				}
				else
				{
					minY = total_min.y();
					maxY = total_center.y();
				}
				if (i & 0b100)
				{
					minZ = total_center.y();
					maxZ = total_max.y();
				}
				else
				{
					minZ = total_min.y();
					maxZ = total_center.y();
				}

				Eigen::Vector3f min(minX, minY, minZ);
				Eigen::Vector3f max(maxX, maxY, maxZ);
				Eigen::Vector3f diff = max - min;
				Eigen::Vector3f center = (max + min) * 0.5f;

				uint3 dimensions;
				dimensions.x = (uint32_t)ceilf(diff.x() / voxelSize);
				dimensions.y = (uint32_t)ceilf(diff.y() / voxelSize);
				dimensions.z = (uint32_t)ceilf(diff.z() / voxelSize);

				size_t numberOfVoxels = dimensions.x * dimensions.y * dimensions.z;

				thrust::for_each(thrust::counting_iterator((size_t)0), thrust::counting_iterator(volume.size()),
					[=] __device__(size_t index) {
					d_volume[index].normal = Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					d_volume[index].weight = 0;
					d_divergences[index] = 0.0f;
					d_potentials[index] = 0.0f;
				});

				{
					t = Time::Now();
					nvtxRangePushA("Insert Points");
					thrust::for_each(
						thrust::make_zip_iterator(thrust::make_tuple(patchBuffers.inputPoints.begin(), patchBuffers.inputNormals.begin())),
						thrust::make_zip_iterator(thrust::make_tuple(patchBuffers.inputPoints.end(), patchBuffers.inputNormals.end())),
						[=] __device__(thrust::tuple<Eigen::Vector3f, Eigen::Vector3f> t) {
						Eigen::Vector3f point = thrust::get<0>(t);
						Eigen::Vector3f normal = thrust::get<1>(t);

						auto index = GetIndex(center, dimensions, voxelSize, point);
						if (index.x == UINT_MAX || index.y == UINT_MAX || index.z == UINT_MAX) return;

						auto flatIndex = GetFlatIndex(index, dimensions);

						atomicAdd(&(d_volume[flatIndex].normal.x()), normal.x());
						atomicAdd(&(d_volume[flatIndex].normal.y()), normal.y());
						atomicAdd(&(d_volume[flatIndex].normal.z()), normal.z());
						atomicAdd(&(d_volume[flatIndex].weight), 1);
					});
					nvtxRangePop();
					t = Time::End(t, "Insert Points");
				}

				hipDeviceSynchronize();

				{
					t = Time::Now();
					nvtxRangePushA("Compute Divergence");
					/*
					thrust::for_each(thrust::counting_iterator((size_t)0), thrust::counting_iterator(volume.size()),
						[=] __device__(size_t index) {
						size_t indexZ = index / (dimensions.y * dimensions.x);
						size_t indexY = (index % (dimensions.y * dimensions.x)) / dimensions.x;
						size_t indexX = (index % (dimensions.y * dimensions.x)) % dimensions.x;

						Voxel cv = d_volume[index];

						size_t piX = (indexX > 0) ? indexX - 1 : indexX;
						size_t niX = (indexX < dimensions.x - 1) ? indexX + 1 : indexX;
						size_t piY = (indexY > 0) ? indexY - 1 : indexY;
						size_t niY = (indexY < dimensions.y - 1) ? indexY + 1 : indexY;
						size_t piZ = (indexZ > 0) ? indexZ - 1 : indexZ;
						size_t niZ = (indexZ < dimensions.z - 1) ? indexZ + 1 : indexZ;

						size_t flatIndexX1 = GetFlatIndex(make_uint3(piX, indexY, indexZ), dimensions);
						size_t flatIndexX2 = GetFlatIndex(make_uint3(niX, indexY, indexZ), dimensions);
						size_t flatIndexY1 = GetFlatIndex(make_uint3(indexX, piY, indexZ), dimensions);
						size_t flatIndexY2 = GetFlatIndex(make_uint3(indexX, niY, indexZ), dimensions);
						size_t flatIndexZ1 = GetFlatIndex(make_uint3(indexX, indexY, piZ), dimensions);
						size_t flatIndexZ2 = GetFlatIndex(make_uint3(indexX, indexY, niZ), dimensions);

						Eigen::Vector3f normX1 = d_volume[flatIndexX1].weight > 0 ? d_volume[flatIndexX1].normal / (float)d_volume[flatIndexX1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
						Eigen::Vector3f normX2 = d_volume[flatIndexX2].weight > 0 ? d_volume[flatIndexX2].normal / (float)d_volume[flatIndexX2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
						Eigen::Vector3f normY1 = d_volume[flatIndexY1].weight > 0 ? d_volume[flatIndexY1].normal / (float)d_volume[flatIndexY1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
						Eigen::Vector3f normY2 = d_volume[flatIndexY2].weight > 0 ? d_volume[flatIndexY2].normal / (float)d_volume[flatIndexY2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
						Eigen::Vector3f normZ1 = d_volume[flatIndexZ1].weight > 0 ? d_volume[flatIndexZ1].normal / (float)d_volume[flatIndexZ1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
						Eigen::Vector3f normZ2 = d_volume[flatIndexZ2].weight > 0 ? d_volume[flatIndexZ2].normal / (float)d_volume[flatIndexZ2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);

						float divX = 0.0f, divY = 0.0f, divZ = 0.0f;

						if (indexX > 0 && indexX < dimensions.x - 1) {
							divX = (normX2.x() - normX1.x()) / (2.0f * voxelSize);
						}
						else if (indexX == 0) {
							divX = (normX2.x() - cv.normal.x()) / voxelSize;
						}
						else if (indexX == dimensions.x - 1) {
							divX = (cv.normal.x() - normX1.x()) / voxelSize;
						}

						if (indexY > 0 && indexY < dimensions.y - 1) {
							divY = (normY2.y() - normY1.y()) / (2.0f * voxelSize);
						}
						else if (indexY == 0) {
							divY = (normY2.y() - cv.normal.y()) / voxelSize;
						}
						else if (indexY == dimensions.y - 1) {
							divY = (cv.normal.y() - normY1.y()) / voxelSize;
						}

						if (indexZ > 0 && indexZ < dimensions.z - 1) {
							divZ = (normZ2.z() - normZ1.z()) / (2.0f * voxelSize);
						}
						else if (indexZ == 0) {
							divZ = (normZ2.z() - cv.normal.z()) / voxelSize;
						}
						else if (indexZ == dimensions.z - 1) {
							divZ = (cv.normal.z() - normZ1.z()) / voxelSize;
						}

						d_divergences[index] = divX + divY + divZ;

						d_volume[index] = cv;
					});
					nvtxRangePop();
					*/

					thrust::for_each(thrust::counting_iterator<size_t>(0), thrust::counting_iterator<size_t>(numberOfVoxels),
						[=] __device__(size_t index) {
						// Skip empty voxels
						if (d_volume[index].weight == 0) {
							d_divergences[index] = 0.0f;
							return;
						}

						// Compute indices
						size_t x = index % dimensions.x;
						size_t y = (index / dimensions.x) % dimensions.y;
						size_t z = index / (dimensions.x * dimensions.y);

						// Compute divergence
						float divX = 0.0f, divY = 0.0f, divZ = 0.0f;

						// X-direction
						if (x == 0) {
							divX = (d_volume[GetFlatIndex(make_uint3(x + 1, y, z), dimensions)].normal.x() -
								d_volume[index].normal.x()) / voxelSize;
						}
						else if (x == dimensions.x - 1) {
							divX = (d_volume[index].normal.x() -
								d_volume[GetFlatIndex(make_uint3(x - 1, y, z), dimensions)].normal.x()) / voxelSize;
						}
						else {
							divX = (d_volume[GetFlatIndex(make_uint3(x + 1, y, z), dimensions)].normal.x() -
								d_volume[GetFlatIndex(make_uint3(x - 1, y, z), dimensions)].normal.x()) / (2.0f * voxelSize);
						}

						// Y-direction
						if (y == 0) {
							divY = (d_volume[GetFlatIndex(make_uint3(x, y + 1, z), dimensions)].normal.y() -
								d_volume[index].normal.y()) / voxelSize;
						}
						else if (y == dimensions.y - 1) {
							divY = (d_volume[index].normal.y() -
								d_volume[GetFlatIndex(make_uint3(x, y - 1, z), dimensions)].normal.y()) / voxelSize;
						}
						else {
							divY = (d_volume[GetFlatIndex(make_uint3(x, y + 1, z), dimensions)].normal.y() -
								d_volume[GetFlatIndex(make_uint3(x, y - 1, z), dimensions)].normal.y()) / (2.0f * voxelSize);
						}

						// Z-direction
						if (z == 0) {
							divZ = (d_volume[GetFlatIndex(make_uint3(x, y, z + 1), dimensions)].normal.z() -
								d_volume[index].normal.z()) / voxelSize;
						}
						else if (z == dimensions.z - 1) {
							divZ = (d_volume[index].normal.z() -
								d_volume[GetFlatIndex(make_uint3(x, y, z - 1), dimensions)].normal.z()) / voxelSize;
						}
						else {
							divZ = (d_volume[GetFlatIndex(make_uint3(x, y, z + 1), dimensions)].normal.z() -
								d_volume[GetFlatIndex(make_uint3(x, y, z - 1), dimensions)].normal.z()) / (2.0f * voxelSize);
						}

						// Set divergence
						d_divergences[index] = divX + divY + divZ;

						// Clamp divergence for stability
						float maxDivergenceThreshold = 100.0f;
						if (fabsf(d_divergences[index]) > maxDivergenceThreshold) {
							d_divergences[index] = copysignf(maxDivergenceThreshold, d_divergences[index]);
						}
					});


					t = Time::End(t, "Compute Divergence");
				}

				hipDeviceSynchronize();

				//{
				//	thrust::for_each(thrust::counting_iterator((size_t)0), thrust::counting_iterator(volume.size()),
				//		[=] __device__(size_t index) {
				//		d_potentials[index] = d_divergences[index];
				//	});
				//}

				{
					t = Time::Now();
					nvtxRangePushA("Compute Potential");

					thrust::for_each(thrust::counting_iterator<size_t>(0), thrust::counting_iterator<size_t>(numberOfVoxels),
						[=] __device__(size_t idx) {
						//printf("%d\n", idx);

						size_t z = idx / (dimensions.x * dimensions.y);
						size_t y = (idx / dimensions.x) % dimensions.y;
						size_t x = idx % dimensions.x;

						//printf("%d %d %d\n", x, y, z);

						if (x == 0 || x == dimensions.x - 1 || y == 0 || y == dimensions.y - 1 || z == 0 || z == dimensions.z - 1) {
							// Dirichlet ��� ����: ��� ���ټ��� 0
							return;
						}

						float divergence = d_divergences[idx];

						float neighborSum = 0.0f;
						neighborSum += d_potentials[idx - 1];
						neighborSum += d_potentials[idx + 1];
						neighborSum += d_potentials[idx - dimensions.x];
						neighborSum += d_potentials[idx + dimensions.x];
						neighborSum += d_potentials[idx - dimensions.x * dimensions.y];
						neighborSum += d_potentials[idx + dimensions.x * dimensions.y];

						d_potentials[idx] = (neighborSum - divergence * voxelSize * voxelSize) / 6.0f;

						//printf("d_potentials[idx] : %f\n", d_potentials[idx]);
					});

					nvtxRangePop();
					t = Time::End(t, "Compute Potential");
				}



				hipDeviceSynchronize();

				//{
				//	// Add cubes where volume value is not zero
				//	nvtxRangePushA("Add Cubes");
				//	thrust::host_vector<Voxel> h_volume = volume; // Copy device vector to host
				//	thrust::host_vector<float> h_divergences = divergences; // Copy device vector to host
				//	thrust::host_vector<float> h_potentials = potentials; // Copy device vector to host

				//	for (uint32_t z = 0; z < dimensions.z; ++z)
				//	{
				//		for (uint32_t y = 0; y < dimensions.y; ++y)
				//		{
				//			for (uint32_t x = 0; x < dimensions.x; ++x)
				//			{
				//				uint3 index = make_uint3(x, y, z);
				//				size_t flatIndex = GetFlatIndex(index, dimensions);
				//				Voxel& voxel = h_volume[flatIndex];
				//				float divergence = h_divergences[flatIndex];

				//				// �߻� ���� ��ȿ���� Ȯ���ϴ� ���� ��ȭ
				//				if (!isnan(divergence) && divergence != FLT_MAX)
				//				{
				//					if (fabsf(divergence) > 0.5f)  // �߻� ���� ���� ���� ���� �ִ� ��쿡�� ȭ��ǥ �߰�
				//					{
				//						Eigen::Vector3f position = GetPosition(center, dimensions, voxelSize, index);
				//						//VD::AddArrow("Divergences", position, voxel.normal, voxelSize, Color4::Red);
				//						//VD::AddCube("Divergences", position, { 0.1f, 0.1f, 0.1f }, {0.0f, 0.0f, 1.0f}, Color4::White);
				//					}
				//				}
				//			}
				//		}
				//	}
				//	nvtxRangePop();
				//	t = Time::End(t, "Add Cubes");
				//}

				{
					::MarchingCubes::MarchingCubesSurfaceExtractor<float> mc(
						d_potentials,
						make_float3(min.x(), min.y(), min.z()),
						make_float3(max.x(), max.y(), max.z()),
						0.1f,
						0.1f);

					auto result = mc.Extract();

					{
						thrust::host_vector<float> h_field(mc.h_internal->numberOfVoxels);
						auto t_field = thrust::raw_pointer_cast(h_field.data());
						hipMemcpy(t_field, mc.h_internal->data, sizeof(float) * mc.h_internal->numberOfVoxels, hipMemcpyDeviceToHost);
						hipDeviceSynchronize();

						for (size_t i = 0; i < h_field.size(); i++)
						{
							auto zIndex = i / (dimensions.x * dimensions.y);
							auto yIndex = (i % (dimensions.x * dimensions.y)) / dimensions.x;
							auto xIndex = (i % (dimensions.x * dimensions.y)) % dimensions.x;

							if (FLT_MAX != h_field[i])
							{
								auto position = GetPosition({ 0.0f, 0.0f, 0.0f }, dimensions, voxelSize, make_uint3(xIndex, yIndex, zIndex));
								VD::AddCube("occupied", position, { 0.05f, 0.05f, 0.05f }, { 0.0f, 0.0f, 1.0f }, Color4::White);
							}
						}
					}

					PLYFormat ply;

					for (size_t i = 0; i < result.numberOfVertices; i++)
					{
						auto v = result.vertices[i];
						ply.AddPoint(v.x, v.y, v.z);
					}

					for (size_t i = 0; i < result.numberOfTriangles; i++)
					{
						auto t = result.triangles[i];
						ply.AddIndex(t.x);
						ply.AddIndex(t.y);
						ply.AddIndex(t.z);
					}

					ply.Serialize("C:\\Resources\\Debug\\Field.ply");

					for (size_t i = 0; i < result.numberOfTriangles; i++)
					{
						auto i0 = result.triangles[i].x;
						auto i1 = result.triangles[i].y;
						auto i2 = result.triangles[i].z;

						auto v0 = result.vertices[i0];
						auto v1 = result.vertices[i1];
						auto v2 = result.vertices[i2];

						VD::AddTriangle("Marching Cubes", { v0.x, v0.y, v0.z }, { v1.x, v1.y, v1.z }, { v2.x, v2.y, v2.z }, Color4::White);
					}

					delete result.vertices;
					delete result.triangles;

					return;
				}
			}
		}
	}
}

#include "hip/hip_runtime.h"
#include "PSR.cuh"

#include <vtkHeaderFiles.h>

#include <App/Utility.h>

#include <Debugging/VisualDebugging.h>
using VD = VisualDebugging;

namespace CUDA
{
	namespace PSR
	{
#pragma region PatchBuffers
		PatchBuffers::PatchBuffers(int width, int height)
			: width(width), height(height),
			inputPoints(thrust::device_vector<Eigen::Vector3f>(width* height)),
			inputNormals(thrust::device_vector<Eigen::Vector3f>(width* height)),
			inputColors(thrust::device_vector<Eigen::Vector3f>(width* height))
		{
		}

		PatchBuffers::~PatchBuffers()
		{
		}

		void PatchBuffers::Clear()
		{
			numberOfInputPoints = 0;
			Eigen::Vector3f zeroVector(0.0f, 0.0f, 0.0f);
			thrust::fill(inputPoints.begin(), inputPoints.end(), zeroVector);
			thrust::fill(inputNormals.begin(), inputNormals.end(), zeroVector);
			thrust::fill(inputColors.begin(), inputColors.end(), zeroVector);
		}

		void PatchBuffers::FromPLYFile(const PLYFormat& ply)
		{
			Clear();

			numberOfInputPoints = ply.GetPoints().size() / 3;

			hipMemcpy(thrust::raw_pointer_cast(inputPoints.data()),
				ply.GetPoints().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			hipMemcpy(thrust::raw_pointer_cast(inputNormals.data()),
				ply.GetPoints().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			hipMemcpy(thrust::raw_pointer_cast(inputColors.data()),
				ply.GetPoints().data(),
				sizeof(Eigen::Vector3f) * numberOfInputPoints,
				hipMemcpyHostToDevice);

			//for (size_t i = 0; i < ply.GetPoints().size() / 3; i++)
			//{
			//	auto px = ply.GetPoints()[i * 3 + 0];
			//	auto py = ply.GetPoints()[i * 3 + 1];
			//	auto pz = ply.GetPoints()[i * 3 + 2];

			//	auto nx = ply.GetNormals()[i * 3 + 0];
			//	auto ny = ply.GetNormals()[i * 3 + 1];
			//	auto nz = ply.GetNormals()[i * 3 + 2];

			//	auto cx = ply.GetColors()[i * 3 + 0];
			//	auto cy = ply.GetColors()[i * 3 + 1];
			//	auto cz = ply.GetColors()[i * 3 + 2];

			//	auto c4 = Color4::FromNormalized(cx, cy, cz, 1.0f);
			//	VD::AddSphere("points", { px, py, pz }, { 0.05f, 0.05f, 0.05f }, { nx, ny, nz }, c4);
			//}
		}
#pragma endregion

		struct Voxel
		{
			Eigen::Vector3f normal;
			int weight;
			float divergence;
			__host__ __device__ Voxel() : normal(0.0f, 0.0f, 0.0f), weight(0), divergence(FLT_MAX) {}
		};

		__host__ __device__
			uint64_t GetMortonCode(
				const Eigen::Vector3f& min,
				const Eigen::Vector3f& max,
				int maxDepth,
				const Eigen::Vector3f& position) {
			// Validate and compute range
			Eigen::Vector3f range = max - min;
			range = range.cwiseMax(Eigen::Vector3f::Constant(1e-6f)); // Avoid zero range

			// Normalize position
			Eigen::Vector3f relativePos = (position - min).cwiseQuotient(range);

			// Clamp to [0, 1]
			relativePos = relativePos.cwiseMax(0.0f).cwiseMin(1.0f);

			// Scale to Morton grid size
			uint32_t maxCoordinateValue = (1 << maxDepth) - 1; // maxCoordinateValue = 1 for maxDepth = 1
			uint32_t x = static_cast<uint32_t>(roundf(relativePos.x() * maxCoordinateValue));
			uint32_t y = static_cast<uint32_t>(roundf(relativePos.y() * maxCoordinateValue));
			uint32_t z = static_cast<uint32_t>(roundf(relativePos.z() * maxCoordinateValue));

			// Compute Morton code
			uint64_t mortonCode = 0;
			for (int i = 0; i < maxDepth; ++i) {
				mortonCode |= ((x >> i) & 1ULL) << (3 * i);
				mortonCode |= ((y >> i) & 1ULL) << (3 * i + 1);
				mortonCode |= ((z >> i) & 1ULL) << (3 * i + 2);
			}

			return mortonCode;
		}

		__host__ __device__
			uint3 GetIndex(const Eigen::Vector3f& gridCenter, uint3 gridDimensions, float voxelSize, const Eigen::Vector3f& position)
		{
			Eigen::Vector3f halfGridSize = Eigen::Vector3f(
				(float)gridDimensions.x * voxelSize * 0.5f,
				(float)gridDimensions.y * voxelSize * 0.5f,
				(float)gridDimensions.z * voxelSize * 0.5f
			);

			Eigen::Vector3f gridMin = gridCenter - halfGridSize;
			Eigen::Vector3f relativePosition = position - gridMin;

			uint3 index = make_uint3(UINT_MAX, UINT_MAX, UINT_MAX);

			if (relativePosition.x() < 0.0f || relativePosition.x() >= (float)gridDimensions.x * voxelSize ||
				relativePosition.y() < 0.0f || relativePosition.y() >= (float)gridDimensions.y * voxelSize ||
				relativePosition.z() < 0.0f || relativePosition.z() >= (float)gridDimensions.z * voxelSize)
			{
				return index;
			}
			else
			{
				index.x = (uint32_t)floorf(relativePosition.x() / voxelSize);
				index.y = (uint32_t)floorf(relativePosition.y() / voxelSize);
				index.z = (uint32_t)floorf(relativePosition.z() / voxelSize);
			}

			return index;
		}

		__host__ __device__
			Eigen::Vector3f GetPosition(const Eigen::Vector3f& gridCenter, uint3 gridDimensions, float voxelSize, const uint3& index)
		{
			Eigen::Vector3f halfGridSize = Eigen::Vector3f(
				(float)gridDimensions.x * voxelSize * 0.5f,
				(float)gridDimensions.y * voxelSize * 0.5f,
				(float)gridDimensions.z * voxelSize * 0.5f
			);

			Eigen::Vector3f gridMin = gridCenter - halfGridSize;

			// Calculate the position of the given voxel using the provided index
			Eigen::Vector3f position = Eigen::Vector3f(
				gridMin.x() + (float)index.x * voxelSize/* + voxelSize * 0.5f*/,
				gridMin.y() + (float)index.y * voxelSize/* + voxelSize * 0.5f*/,
				gridMin.z() + (float)index.z * voxelSize/* + voxelSize * 0.5f*/
			);

			return position;
		}

		__host__ __device__
			size_t GetFlatIndex(const uint3& index, const uint3& dimensions)
		{
			return index.z * dimensions.x * dimensions.y + index.y * dimensions.x + index.x;
		}

		__host__ __device__ bool ShouldAddCube(const Voxel& voxel) {
			if (voxel.divergence == FLT_MAX) {
				return false;
			}
			return fabsf(voxel.divergence) <= 1.0f;
		}

		void TestPSR()
		{
			auto t = Time::Now();

			stringstream ss;
			ss << "C:\\Resources\\3D\\PLY\\Complete\\Lower_pointcloud.ply";

			PLYFormat ply;
			ply.Deserialize(ss.str());
			cout << "ply min : " << ply.GetAABB().min().transpose() << endl;
			cout << "ply max : " << ply.GetAABB().max().transpose() << endl;

			t = Time::End(t, "Load ply");

			PatchBuffers patchBuffers(ply.GetPoints().size() / 3, 1);
			patchBuffers.FromPLYFile(ply);

			t = Time::End(t, "Copy data to device");

			Eigen::Vector3f min(-10.0f, -10.0f, -10.0f);
			Eigen::Vector3f max(10.0f, 10.0f, 10.0f);
			//Eigen::Vector3f min(-75.0f, -75.0f, -75.0f);
			//Eigen::Vector3f max(75.0f, 75.0f, 75.0f);
			Eigen::Vector3f diff = max - min;
			Eigen::Vector3f center = (max + min) * 0.5f;
			float voxelSize = 0.1f;
			uint3 dimensions;
			dimensions.x = (uint32_t)ceilf(diff.x() / voxelSize);
			dimensions.y = (uint32_t)ceilf(diff.y() / voxelSize);
			dimensions.z = (uint32_t)ceilf(diff.z() / voxelSize);
			

			thrust::device_vector<Voxel> volume(dimensions.x * dimensions.y * dimensions.z);
			Voxel defaultVoxel;
			thrust::fill_n(volume.begin(), dimensions.x * dimensions.y * dimensions.z, defaultVoxel);
			auto d_volume = thrust::raw_pointer_cast(volume.data());

			{
				t = Time::Now();
				nvtxRangePushA("Insert Points");
				thrust::for_each(
					thrust::make_zip_iterator(thrust::make_tuple(patchBuffers.inputPoints.begin(), patchBuffers.inputNormals.begin())),
					thrust::make_zip_iterator(thrust::make_tuple(patchBuffers.inputPoints.end(), patchBuffers.inputNormals.end())),
					[=] __device__(thrust::tuple<Eigen::Vector3f, Eigen::Vector3f> t) {
					Eigen::Vector3f point = thrust::get<0>(t);
					Eigen::Vector3f normal = thrust::get<1>(t);

					auto index = GetIndex(center, dimensions, voxelSize, point);
					if (index.x == UINT_MAX || index.y == UINT_MAX || index.z == UINT_MAX) return;

					auto flatIndex = GetFlatIndex(index, dimensions);

					atomicAdd(&(d_volume[flatIndex].normal.x()), normal.x());
					atomicAdd(&(d_volume[flatIndex].normal.y()), normal.y());
					atomicAdd(&(d_volume[flatIndex].normal.z()), normal.z());
					atomicAdd(&(d_volume[flatIndex].weight), 1);
				});
				nvtxRangePop();
				t = Time::End(t, "Insert Points");
			}

			{
				t = Time::Now();
				nvtxRangePushA("Compute Divergence");
				thrust::for_each(thrust::counting_iterator((size_t)0), thrust::counting_iterator(volume.size()),
					[=] __device__(size_t index) {
					size_t indexZ = index / (dimensions.y * dimensions.x);
					size_t indexY = (index % (dimensions.y * dimensions.x)) / dimensions.x;
					size_t indexX = (index % (dimensions.y * dimensions.x)) % dimensions.x;

					Voxel cv = d_volume[index];  // ���� ���� �� ����

					// ��� ���ǿ� ���� ����/���� �ε��� ���
					size_t piX = (indexX > 0) ? indexX - 1 : indexX;
					size_t niX = (indexX < dimensions.x - 1) ? indexX + 1 : indexX;
					size_t piY = (indexY > 0) ? indexY - 1 : indexY;
					size_t niY = (indexY < dimensions.y - 1) ? indexY + 1 : indexY;
					size_t piZ = (indexZ > 0) ? indexZ - 1 : indexZ;
					size_t niZ = (indexZ < dimensions.z - 1) ? indexZ + 1 : indexZ;

					// ���� ������ ���� ������ �ε����� ������ ��������
					size_t flatIndexX1 = GetFlatIndex(make_uint3(piX, indexY, indexZ), dimensions);
					size_t flatIndexX2 = GetFlatIndex(make_uint3(niX, indexY, indexZ), dimensions);
					size_t flatIndexY1 = GetFlatIndex(make_uint3(indexX, piY, indexZ), dimensions);
					size_t flatIndexY2 = GetFlatIndex(make_uint3(indexX, niY, indexZ), dimensions);
					size_t flatIndexZ1 = GetFlatIndex(make_uint3(indexX, indexY, piZ), dimensions);
					size_t flatIndexZ2 = GetFlatIndex(make_uint3(indexX, indexY, niZ), dimensions);

					// �̿� ������ ��� ���Ϳ� ����ġ�� Ȯ���Ͽ� ��ȿ�� ���� ���
					Eigen::Vector3f normX1 = d_volume[flatIndexX1].weight > 0 ? d_volume[flatIndexX1].normal / (float)d_volume[flatIndexX1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					Eigen::Vector3f normX2 = d_volume[flatIndexX2].weight > 0 ? d_volume[flatIndexX2].normal / (float)d_volume[flatIndexX2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					Eigen::Vector3f normY1 = d_volume[flatIndexY1].weight > 0 ? d_volume[flatIndexY1].normal / (float)d_volume[flatIndexY1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					Eigen::Vector3f normY2 = d_volume[flatIndexY2].weight > 0 ? d_volume[flatIndexY2].normal / (float)d_volume[flatIndexY2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					Eigen::Vector3f normZ1 = d_volume[flatIndexZ1].weight > 0 ? d_volume[flatIndexZ1].normal / (float)d_volume[flatIndexZ1].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);
					Eigen::Vector3f normZ2 = d_volume[flatIndexZ2].weight > 0 ? d_volume[flatIndexZ2].normal / (float)d_volume[flatIndexZ2].weight : Eigen::Vector3f(0.0f, 0.0f, 0.0f);

					// �߻� ��� (�߽� ���� ���)
					float divX = (normX2.x() - normX1.x()) / (2.0f * voxelSize);
					float divY = (normY2.y() - normY1.y()) / (2.0f * voxelSize);
					float divZ = (normZ2.z() - normZ1.z()) / (2.0f * voxelSize);

					// �߻� ����� ���� ������ ����
					cv.divergence = divX + divY + divZ;

					// ����� �ٽ� d_volume�� ����
					d_volume[index] = cv;
				});
				nvtxRangePop();
				t = Time::End(t, "Compute Divergence");
			}

			{
				// Add cubes where volume value is not zero
				nvtxRangePushA("Add Cubes");
				thrust::host_vector<Voxel> h_volume = volume; // Copy device vector to host
				for (uint32_t z = 0; z < dimensions.z; ++z)
				{
					for (uint32_t y = 0; y < dimensions.y; ++y)
					{
						for (uint32_t x = 0; x < dimensions.x; ++x)
						{
							uint3 index = make_uint3(x, y, z);
							size_t flatIndex = GetFlatIndex(index, dimensions);
							Voxel& voxel = h_volume[flatIndex];

							// �߻� ���� ��ȿ���� Ȯ���ϴ� ���� ��ȭ
							if (!isnan(voxel.divergence) && voxel.divergence != FLT_MAX)
							{
								if (fabsf(voxel.divergence) > 0.0f)  // �߻� ���� ���� ���� ���� �ִ� ��쿡�� ť�� �߰�
								{
									Eigen::Vector3f position = GetPosition(center, dimensions, voxelSize, index);
									VD::AddCube("volume", position, { voxelSize, voxelSize, voxelSize },
										{ 0.0f, 0.0f, 1.0f }, Color4::White);
								}
							}
						}
					}
				}
				nvtxRangePop();
				t = Time::End(t, "Add Cubes");
			}
		}
	}
}
